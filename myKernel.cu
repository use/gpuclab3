#include "hip/hip_runtime.h"
#include "myKernel.h"

__global__ void kernel( int *a, int dimx, int dimy )
{
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;

    a[idx]  = a[idx]+1;
}

// Please implement the following kernels2 through kernel6,
// in order to meet the requirements in the write-ups. 
__global__ void kernel2( int *a, int dimx, int dimy )
{
}

__global__ void kernel3( int *a, int dimx, int dimy )
{
}

__global__ void kernel4( int *a, int dimx, int dimy )
{
}

__global__ void kernel5( int *a, int dimx, int dimy )
{
}

__global__ void kernel6( int *a, int dimx, int dimy )
{
}


