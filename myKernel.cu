#include "hip/hip_runtime.h"
#include "myKernel.h"

__global__ void kernel( int *a, int dimx, int dimy )
{
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;

    a[idx]  = a[idx]+1;
}

// Please implement the following kernels2 through kernel6,
// in order to meet the requirements in the write-ups. 
__global__ void kernel2( int *a, int dimx, int dimy )
{
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;
    if (ix < dimx && iy < dimy){
        a[idx] = blockIdx.x + blockIdx.y * (dimx / blockDim.x + 1);
    }
}

__global__ void kernel3( int *a, int dimx, int dimy )
{
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;
    if (ix < dimx && iy < dimy){
        a[idx] = idx;
    }
}

__global__ void kernel4( int *a, int dimx, int dimy )
{
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;
    if (ix < dimx && iy < dimy){
        a[idx] = threadIdx.x + threadIdx.y * blockDim.x;
    }
}

__global__ void kernel5( int *a, int dimx, int dimy )
{
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;
    if (ix < dimx && iy < dimy){
        a[idx] = blockIdx.y;
    }
}

__global__ void kernel6( int *a, int dimx, int dimy )
{
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;
    if (ix < dimx && iy < dimy){
        a[idx] = blockIdx.x;
    }
}


