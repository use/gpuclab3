#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "myKernel.h"

void printArr(int *a, char *name, int dimy, int dimx);


int main()
{
    int dimx = 16;
    int dimy = 16;
    int num_bytes = dimx*dimy*sizeof(int);

    int *d_a=0, *h_a=0; // device and host pointers

    h_a = (int*)malloc(num_bytes);
    hipMalloc( (void**)&d_a, num_bytes );

    if( 0==h_a || 0==d_a )
    {
        printf("couldn't allocate memory\n");
        return 1;
    }

    hipMemset( d_a, 0, num_bytes );

    dim3 grid, block;
    block.x = 3;
    block.y = 4;
    grid.x  = ceil( (float)dimx / block.x );
    grid.y  = ceil( (float)dimy / block.y );
    
    // Use kernel to fill d_a array
    kernel<<<grid, block>>>( d_a, dimx, dimy );
    hipMemcpy( h_a, d_a, num_bytes, hipMemcpyDeviceToHost );
    printArr(h_a, "kernel ", dimy, dimx);

    // Use kernel2 to fill d_a array
    hipMemset( d_a, 0, num_bytes );
    kernel2<<<grid, block>>>( d_a, dimx, dimy );
    hipMemcpy( h_a, d_a, num_bytes, hipMemcpyDeviceToHost );
    printArr(h_a, "kernel 2", dimy, dimx);

    // Use kernel3 to fill d_a array
    hipMemset( d_a, 0, num_bytes );
    kernel3<<<grid, block>>>( d_a, dimx, dimy );
    hipMemcpy( h_a, d_a, num_bytes, hipMemcpyDeviceToHost );
    printArr(h_a, "kernel 3", dimy, dimx);

    // Use kernel4 to fill d_a array
    hipMemset( d_a, 0, num_bytes );
    kernel4<<<grid, block>>>( d_a, dimx, dimy );
    hipMemcpy( h_a, d_a, num_bytes, hipMemcpyDeviceToHost );
    printArr(h_a, "kernel 4", dimy, dimx);
   
    // Use kernel5 to fill d_a array
    hipMemset( d_a, 0, num_bytes );
    kernel5<<<grid, block>>>( d_a, dimx, dimy );
    hipMemcpy( h_a, d_a, num_bytes, hipMemcpyDeviceToHost );
    printArr(h_a, "kernel 5", dimy, dimx);
   
    // Use kernel6 to fill d_a array
    hipMemset( d_a, 0, num_bytes );
    kernel6<<<grid, block>>>( d_a, dimx, dimy );
    hipMemcpy( h_a, d_a, num_bytes, hipMemcpyDeviceToHost );
    printArr(h_a, "kernel 6", dimy, dimx);

    free( h_a );
    hipFree( d_a );

    return 0;
}

// print array 
void printArr(int *a, char *name, int dimy, int dimx)
{
    if(name == NULL)
        return;
    printf("===================================%s====================================\n", name);
    for(int row=0; row<dimy; row++)
    {
        for(int col=0; col<dimx; col++)
            printf("%-4d ", a[row*dimx+col] );
        printf("\n");
    }
    printf("==============================================================================\n");
}


